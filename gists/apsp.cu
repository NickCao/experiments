#include "hip/hip_runtime.h"
#include "apsp.h"

const int b = 32;

namespace {
__global__ void kernel1(int n, int *graph, int p) {
  __shared__ int block[b * b];
  auto i = p * b + threadIdx.y;
  auto j = p * b + threadIdx.x;
  auto flag = i < n && j < n;
  block[threadIdx.y * b + threadIdx.x] = flag ? graph[i * n + j] : 10001;
  __syncthreads();
  if (flag) {
    for (int k = 0; k < b; k++) {
      block[threadIdx.y * b + threadIdx.x] =
          min(block[threadIdx.y * b + threadIdx.x],
              block[threadIdx.y * b + k] + block[k * b + threadIdx.x]);
    }
    graph[i * n + j] = block[threadIdx.y * b + threadIdx.x];
  }
}

__global__ void kernel2(int n, int *graph, int p) {
  __shared__ int block[b * b];
  __shared__ int center[b * b];
  auto i = p * b + threadIdx.y;
  auto j = p * b + threadIdx.x;
  auto ti = blockIdx.y * blockDim.y + threadIdx.y;
  auto tj = blockIdx.x * blockDim.x + threadIdx.x;
  bool flag = (blockIdx.x == p || blockIdx.y == p) &&
              !(blockIdx.x == p && blockIdx.y == p);
  if (flag) {
    if (i < n && j < n)
      center[threadIdx.y * b + threadIdx.x] = graph[i * n + j];
    else
      center[threadIdx.y * b + threadIdx.x] = 10001;
    if (ti < n && tj < n)
      block[threadIdx.y * b + threadIdx.x] = graph[ti * n + tj];
    else
      flag = false;
  }
  __syncthreads();
  if (flag) {
    if (blockIdx.x == p) {
      for (int k = 0; k < b; k++) {
        block[threadIdx.y * b + threadIdx.x] =
            min(block[threadIdx.y * b + threadIdx.x],
                block[threadIdx.y * b + k] + center[k * b + threadIdx.x]);
      }
    } else {
      for (int k = 0; k < b; k++) {
        block[threadIdx.y * b + threadIdx.x] =
            min(block[threadIdx.y * b + threadIdx.x],
                center[threadIdx.y * b + k] + block[k * b + threadIdx.x]);
      }
    }
    graph[ti * n + tj] = block[threadIdx.y * b + threadIdx.x];
  }
}

__global__ void kernel3(int n, int *graph, int p) {
  __shared__ int crossx[b * b];
  __shared__ int crossy[b * b];
  auto ix = p * b + threadIdx.y;
  auto jx = blockIdx.x * blockDim.x + threadIdx.x;
  auto iy = blockIdx.y * blockDim.y + threadIdx.y;
  auto jy = p * b + threadIdx.x;
  auto i = blockIdx.y * blockDim.y + threadIdx.y;
  auto j = blockIdx.x * blockDim.x + threadIdx.x;
  int me;
  bool flag = (blockIdx.x != p && blockIdx.y != p);
  if (flag) {
    if (ix < n && jx < n) {
      crossx[threadIdx.y * b + threadIdx.x] = graph[ix * n + jx];
    } else {
      crossx[threadIdx.y * b + threadIdx.x] = 10001;
    }
    if (iy < n && jy < n) {
      crossy[threadIdx.y * b + threadIdx.x] = graph[iy * n + jy];
    } else {
      crossy[threadIdx.y * b + threadIdx.x] = 10001;
    }
    if (i < n && j < n) {
      me = graph[i * n + j];
    } else {
      flag = false;
    }
  }
  __syncthreads();
  if (flag) {
    for (int k = 0; k < b; k++) {
      me = min(me, crossy[threadIdx.y * b + k] + crossx[k * b + threadIdx.x]);
    }
    graph[i * n + j] = me;
  }
}
} // namespace

void apsp(int n, int *graph) {
  int blocks = (n - 1) / b + 1;
  for (int p = 0; p < blocks; p++) {
    dim3 thr(b, b);
    dim3 blk0(1, 1);
    kernel1<<<blk0, thr>>>(n, graph, p);
    dim3 blk1(blocks, blocks);
    kernel2<<<blk1, thr>>>(n, graph, p);
    kernel3<<<blk1, thr>>>(n, graph, p);
  }
}
